#include "hip/hip_runtime.h"
----------------------------967901351434666784066238
Content-Disposition: form-data; name="file"; filename="peer.cu"
Content-Type: application/cu-seeme

// THis change was made for absolute no reason to change this code....i mean its a comment alright....im testing azure function rn

#include <iostream>
#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start=0){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}
__global__ void init(char *ptr){
	ptr[0]='A';
	ptr[1]='B';
}
__global__ void  print(char *ptr){
	printf("ptr[0]=%c ptr[1]=%c \n" , ptr[0],	ptr[1]);
}

int main(){
  /*hipSetDevice(0);
  hipDeviceEnablePeerAccess(1, 0);
  hipSetDevice(1);
  hipDeviceEnablePeerAccess(0, 0);*/
  size_t nbytes = 32768*1024;
  char* src0; // Memory on device 0
  hipSetDevice(0);
  hipMalloc(&src0, nbytes);
  init<<<1,1>>>(src0);
  hipDeviceSynchronize();
  char* dst1; // Memory on device 1
  hipSetDevice(1);
  hipMalloc(&dst1, nbytes);
  print<<<1,1>>>(dst1);
  hipDeviceSynchronize();
  hipSetDevice(0);
 if( hipMemcpyPeerAsync(dst1, 1, src0, 0, nbytes)!=hipSuccess)printf("memcpy error\n");
  hipDeviceSynchronize();
  hipSetDevice(1);
  print<<<1,1>>>(dst1);
  hipDeviceSynchronize();

}

----------------------------967901351434666784066238--
